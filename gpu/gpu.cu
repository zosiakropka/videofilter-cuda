#include "hip/hip_runtime.h"
#include "../headers/gpu.h"

namespace host {

  void alloc_mem_uchar(uchar** array, uint count) {
    (*array) = (uchar*) malloc(count * sizeof (uchar));
    memset((*array), 0, count * sizeof (uchar));
  }

  void alloc_mem_int(int** array, uint count) {
    (*array) = (int*) malloc(count * sizeof (int));
    memset((*array), 0, count * sizeof (int));
  }

  void free_mem_uchar(uchar** array) {
    free((*array));
    (*array) = NULL;
  }

  void free_mem_int(int** array) {
    free((*array));
    (*array) = NULL;
  }

  void cuda_host2dev(const void *host_array, void *dev_array, uint count, size_t size) {
    hipMemcpy(dev_array, host_array, (count * size), hipMemcpyHostToDevice);
  }

}
namespace dev {

  void alloc_mem_uchar(uchar** array, uint count) {
    test(hipMalloc(array, count * sizeof (uchar)));
  }

  void alloc_mem_int(int** array, uint count) {
    test(hipMalloc(array, count * sizeof (int)));
  }

  void free_mem_uchar(uchar** array) {
    test(hipFree((*array)));
    (*array) = NULL;
  }

  void free_mem_int(int** array) {
    test(hipFree((*array)));
    (*array) = NULL;
  }

  void cuda_dev2host(void* dev_array, void* host_array, uint count, size_t size) {
    test(hipMemcpy(host_array, dev_array, (count * size), hipMemcpyDeviceToHost));
  }

  void test(hipError_t result) {
    if (result != hipSuccess) {
      std::cerr << "Error: " << hipGetErrorString(result) << std::endl;
      exit(1);
    }
  }

  hipDeviceProp_t get_prop() {
    hipDeviceProp_t properties;
    test(hipGetDeviceProperties(&properties, 0));
    return properties;
  }

  uint get_max_block_size() {
    return sqrt(get_prop().maxThreadsPerBlock);
  }

  uint get_block_size(uint threads, uint problem) {

    uint max_size = dev::get_max_block_size();
    uint size = sqrt(problem / threads);

    if (size * size > max_size) {
      std::cerr << "Not enough threads." << std::endl;
      return max_size;
    } else if (!size) {
      std::cerr << "Too many threads." << std::endl;
      return max_size;
    }
    return size;
  }

}